#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include <chrono>

using namespace std;

#define M 256
#define N 256
#define K 256
#define BLOCK_SIZE 256 

typedef std::chrono::high_resolution_clock Time;
typedef std::chrono::milliseconds ms;

void init_matrix(float *mat, int rows,int cols) {
    std::random_device dev;
    std::mt19937 rng(128);
    std::uniform_int_distribution<std::mt19937::result_type> dist(1,rows*cols);

    for(int i = 0 ; i < rows*cols; i++) {
        mat[i] = (float)dist(rng);
    }
}

void matmul_cpu(float *A,float *B, float *C,int m,int n,int k) {
    for (int i = 0; i < m;i++) {
        for (int j = 0; j < k; j++) {
            float sum = 0.0f;
            for (int l = 0; l < n; l++ ) {
                sum += A[n * i + l] * B[k * l + j];
             }
            C[i * m + j] = sum;
        }
    }
}

__global__ void matmul_gpu(float *A,float *B, float *C,int m,int n,int k) {
    auto row = blockIdx.y * blockDim.y + threadIdx.y;
    auto col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < m && col < k) {
        float sum = 0.0f;
        for(int l = 0; l < n; l++) {
            sum += A[row * k + l] * B[k * l +  col];
        }
        C[row * n + col] = sum;
    }

}

int main() {

    float *d_a, *d_b, *d_c;
    float *h_a, *h_b, *h_c_cpu, *h_c_gpu;

    size_t size_A = M * N * sizeof(float);
    size_t size_B = N * K * sizeof(float);
    size_t size_C = M * K * sizeof(float);

    h_a = (float*)malloc(size_A);
    h_b = (float*)malloc(size_B);
    h_c_cpu = (float*)malloc(size_C);
    h_c_gpu = (float*)malloc(size_C);
    
    init_matrix(h_a,M,N);
    init_matrix(h_b,N,K);

    hipMalloc(&d_a,size_A);
    hipMalloc(&d_b,size_B);
    hipMalloc(&d_c,size_C);

    hipMemcpy(d_a,h_a,size_A,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,size_B,hipMemcpyDeviceToHost);

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((M + BLOCK_SIZE - 1) / BLOCK_SIZE, (K + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Warm up
    for (int i = 0; i < 3; i++) {
        matmul_cpu(h_a, h_b, h_c_cpu, M, K, N);
        matmul_gpu<<<gridDim, blockDim>>>(d_a, d_b, d_c, M, N, K);
        hipDeviceSynchronize();
    }

   double cpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        auto cpu_start = Time::now();
        matmul_cpu(h_a, h_b, h_c_cpu, M, K, N);
        auto cpu_stop = Time::now();
        std::chrono::duration<double,std::milli> cpu_perf = cpu_stop - cpu_start;
        cpu_total_time += cpu_perf.count();
    }
    double cpu_avg_time = cpu_total_time / 20.0;


    double gpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        auto gpu_start = Time::now();
        matmul_gpu<<<gridDim, blockDim>>>(d_a, d_b, d_c, M, K, N);
        hipDeviceSynchronize();
        auto gpu_stop = Time::now();
        std::chrono::duration<float,std::milli> gpu_perf = gpu_stop - gpu_start;
        gpu_total_time += gpu_perf.count();
    }
    double gpu_avg_time = gpu_total_time / 20.0;

    printf("CPU average time: %f microseconds\n", (cpu_avg_time * 1e6f));
    printf("GPU average time: %f microseconds\n", (gpu_avg_time * 1e6f));

        // Free memory
    free(h_a);
    free(h_b);
    free(h_c_cpu);
    free(h_c_gpu);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);


    return 0;
}