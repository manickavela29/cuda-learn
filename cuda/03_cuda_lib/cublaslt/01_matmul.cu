#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include <hipblaslt.h>

#include <iostream>
#include <vector>
#include <random>
#include <functional>

#define M 4096
#define K 1024
#define N 4096


using namespace std;

// Naive CUDA kernel for matrix multiplication
__global__ void naiveMatrixMultiply(const float* A, const float* B, float* C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k; ++i) {
            sum += A[row * k + i] * B[i * n + col];
        }
        C[row * n + col] = sum;
    }
}

void init_matrix(float *mat, int rows,int cols) {
    std::random_device dev;
    std::mt19937 rng(128);
    std::uniform_real_distribution<float> dist(-0.5,0.5);

    for(int i = 0 ; i < rows*cols; i++) {
        mat[i] = static_cast<float>(dist(rng));
    }
}

float time_kernel(std::function<void()> kernel_func) {
    hipEvent_t start, stop;
    float elapsed_time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    kernel_func();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsed_time, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsed_time;
}

float benchmark_kernel(std::function<void()> kernel_func,int warmup, int benchrun) {

    //Warm up
    for(int i = 0; i < warmup; i++) {
        kernel_func();
    }

    vector<float> times;
    //benchamrk
    for(int i = 0; i < benchrun; i++) {
        float time = time_kernel(kernel_func);
        times.push_back(time);
    }

    float avg_time = std::accumulate(times.begin(),times.end(),0.0f) / benchrun;
    return avg_time;
}

bool verifyResults(const vector<float>& expected, const vector<float>& actual, float tolerance = 1e-2) {
    if (expected.size() != actual.size()) {
        return false;
    }
    for (size_t i = 0; i < expected.size(); ++i) {
        float rel_error = std::abs(expected[i] - actual[i]);
        if (rel_error > tolerance) {
            std::cout << "Mismatch at index " << i << ": expected " << expected[i] 
                      << ", got " << actual[i] << ", relative error: " << rel_error << std::endl;
            return false;
        }
    }
    return true;
}

int main() {

    vector<float> h_A(M * K), h_B(K * N), h_C(M * N);
    vector<float> h_C_cublas_fp32(M * N), h_C_cublasLt_fp32(M * N);
    vector<float> h_C_cublas_fp16(M * N), h_C_cublasLt_fp16(M * N);
    vector<float> h_C_naive(M * N);
    vector<half> h_C_half(M * N);

    init_matrix(h_A.data(), M, K);
    init_matrix(h_B.data(), K, N);

    float *d_A, *d_B, *d_C;
    half *d_A_half, *d_B_half, *d_C_half;

    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));
    hipMalloc(&d_A_half, M * K * sizeof(half));
    hipMalloc(&d_B_half, K * N * sizeof(half));
    hipMalloc(&d_C_half, M * N * sizeof(half));

    hipMemcpy(d_A,h_A.data(), M * K * sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B.data(), K * N * sizeof(float),hipMemcpyHostToDevice);

    // Handling Half precision
    vector<half> h_A_half(M * K), h_B_half(K * N);
    for(int i = 0; i < M * K; i++) h_A_half[i] = __float2half(h_A[i]);
    for(int i = 0; i < K * N; i++) h_B_half[i] = __float2half(h_B[i]);

    hipMemcpy(d_A_half, h_A_half.data(), M * K * sizeof(half),hipMemcpyHostToDevice);
    hipMemcpy(d_B_half, h_B_half.data(), K * N * sizeof(half),hipMemcpyHostToDevice);

    const int warmup = 3, benchrun = 10;

    //Benchmarking Native CUDA Matmul

    dim3 blockDim(32,32);
    dim3 gridDim((N + blockDim.x - 1)/blockDim.x, (M + blockDim.y -1)/blockDim.y);

    float naive_cuda_time = benchmark_kernel([&]() {
        naiveMatrixMultiply<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);
    }, warmup, benchrun);

    hipMemcpy(h_C_naive.data(), d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // Benchamrking with cublas
    hipblasHandle_t cublas_handle;
    hipblasLtHandle_t cublasLt_handle;

    hipblasCreate(&cublas_handle);
    hipblasLtCreate(&cublasLt_handle);

    float alpha = 1.0f, beta = 0.0f;
    half alpha_half = __float2half(1.0), beta_half = __float2half(0.0f);

    float cublas_fp32_time = benchmark_kernel([&]() {
        hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N);
    }, warmup, benchrun);

    float cublas_fp16_time = benchmark_kernel([&]() {
        hipblasHgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha_half, d_B_half, N, d_A_half, K, &beta_half, d_C_half, N);
    }, warmup, benchrun);
    hipMemcpy(h_C_half.data(), d_C_half, M * N * sizeof(half), hipMemcpyDeviceToHost);
    for(int i = 0; i < M * N; i++) h_C_cublas_fp16[i] = __half2float(h_C_half[i]);

    hipMemcpy(h_C_cublas_fp32.data(), d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
    // cudaMemcpy(h_C_cublas_fp16.data(), d_C_half, K * N * sizeof(half), cudaMemcpyDeviceToHost);

    // Becnhamrkaing with cublasLt
    hipblasLtMatmulDesc_t operationDesc = nullptr;
    hipblasLtMatrixLayout_t Adesc = nullptr, Bdesc = nullptr, Cdesc = nullptr;
    hipblasLtMatmulDescCreate(&operationDesc,HIPBLAS_COMPUTE_32F, HIP_R_32F);
    hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, K, M, K);
    hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, N, K, N);
    hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, N, M, N);

    hipblasLtMatmulDesc_t operationDesc_half = nullptr;
    hipblasLtMatrixLayout_t Adesc_half = nullptr, Bdesc_half = nullptr, Cdesc_half = nullptr;
    hipblasLtMatmulDescCreate(&operationDesc_half, HIPBLAS_COMPUTE_16F, HIP_R_16F);
    hipblasLtMatrixLayoutCreate(&Adesc_half, HIP_R_16F, K, M, K);
    hipblasLtMatrixLayoutCreate(&Bdesc_half, HIP_R_16F, N, K, N);
    hipblasLtMatrixLayoutCreate(&Cdesc_half, HIP_R_16F, N, M, N);

    float cublasLt_fp32_time = benchmark_kernel([&]() {
        hipblasLtMatmul(cublasLt_handle, operationDesc, &alpha, d_B, Bdesc, d_A, Adesc, &beta, d_C, Cdesc, d_C, Cdesc, nullptr, nullptr, 0, 0);
    }, warmup, benchrun);

    float cublasLt_fp16_time = benchmark_kernel([&]() {
        hipblasLtMatmul(cublasLt_handle, operationDesc_half, &alpha_half, d_B_half, Bdesc_half, d_A_half, Adesc_half, &beta_half, d_C_half, Cdesc_half, d_C_half, Cdesc_half, nullptr, nullptr, 0, 0);
    }, warmup, benchrun);

    hipMemcpy(h_C_cublasLt_fp32.data(), d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_C_half.data(), d_C_half, M * N * sizeof(half), hipMemcpyDeviceToHost);
    for(int i = 0; i < M * N; i++) h_C_cublasLt_fp16[i] = __half2float(h_C_half[i]);

    float max_error_fp16_cublas = 0.0f;
    float max_error_fp16_cublasLt = 0.0f;
    float max_error_fp32_cublas = 0.0f;
    float max_error_fp32_cublasLt = 0.0f;

    for(int i = 0 ; i < M * N ; i++) {
        float error = std::abs(h_C_naive[i] - h_C_cublas_fp16[i]);
        if(error > max_error_fp16_cublas) {
            max_error_fp16_cublas = error;
        }

        error = std::abs(h_C_naive[i] - h_C_cublasLt_fp16[i]);
        if(error > max_error_fp16_cublasLt) {
            max_error_fp16_cublasLt = error;
        }

        error = std::abs(h_C_naive[i] - h_C_cublas_fp32[i]);
        if(error > max_error_fp32_cublas) {
            max_error_fp32_cublas = error;
        }

        error = std::abs(h_C_naive[i] - h_C_cublasLt_fp32[i]);
        if(error > max_error_fp32_cublasLt) {
            max_error_fp32_cublasLt = error;
        }
    }


    cout <<"\nPerformance Testing" << endl;
    cout <<"================\n" << endl;

    cout <<"Naive CUDA matmul    : " << naive_cuda_time << endl;
    cout <<"cuBlAS FP32 matmul   : " << cublas_fp32_time << endl;
    cout <<"cuBlASLt FP32 matmul : " << cublasLt_fp32_time << endl;
    cout <<"cuBlAS FP16 matmul   : " << cublas_fp16_time << endl;
    cout <<"cuBlASLt FP16 matmul : " << cublasLt_fp16_time << endl;

    cout <<"\n\nAccuracy Testing" << endl;
    cout <<"================\n" << endl;

    cout << "cuBLAS max fp16 error : " << max_error_fp16_cublas << endl;
    cout << "cuBLASLT max fp16 error : " << max_error_fp16_cublasLt << endl;
    cout << "cuBLAS max fp32 error : " << max_error_fp32_cublas << endl;
    cout << "cuBLASLT max fp32 error : " << max_error_fp32_cublasLt << endl;

    bool cublas_fp32_correct = verifyResults(h_C_naive,h_C_cublas_fp32, 1e-2);
    bool cublasLt_fp32_correct = verifyResults(h_C_naive, h_C_cublasLt_fp32, 1e-2);
    bool cublas_fp16_correct = verifyResults(h_C_naive, h_C_cublas_fp16, 5e-1);
    bool cublasLt_fp16_correct = verifyResults(h_C_naive, h_C_cublasLt_fp16, 5e-1);

    std::cout << "cuBLAS FP32 results " << (cublas_fp32_correct ? "match" : "do not match") << " the naive kernel results within tolerance of 1e-2." << std::endl;
    std::cout << "cuBLASLt FP32 results " << (cublasLt_fp32_correct ? "match" : "do not match") << " the naive kernel results within tolerance of 1e-2." << std::endl;
    std::cout << "cuBLAS FP16 results " << (cublas_fp16_correct ? "match" : "do not match") << " the naive kernel results within tolerance of 5e-1." << std::endl;
    std::cout << "cuBLASLt FP16 results " << (cublasLt_fp16_correct ? "match" : "do not match") << " the naive kernel results within tolerance of 5e-1." << std::endl;

    hipblasLtMatmulDescDestroy(operationDesc);
    hipblasLtMatrixLayoutDestroy(Adesc);
    hipblasLtMatrixLayoutDestroy(Bdesc);
    hipblasLtMatrixLayoutDestroy(Cdesc);
    hipblasLtMatmulDescDestroy(operationDesc_half);
    hipblasLtMatrixLayoutDestroy(Adesc_half);
    hipblasLtMatrixLayoutDestroy(Bdesc_half);
    hipblasLtMatrixLayoutDestroy(Cdesc_half);
    hipblasLtDestroy(cublasLt_handle);
    hipblasDestroy(cublas_handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_A_half);
    hipFree(d_B_half);
    hipFree(d_C_half);

    return 0;
}

